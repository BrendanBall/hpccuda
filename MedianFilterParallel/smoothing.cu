#include "hip/hip_runtime.h"
#include "smoothing.h"
#include <iostream>
#include <algorithm>
#include <math.h> 
#include <cstring>
#include "hip/hip_runtime.h"
#include "hip/device_functions.h"
#include ""
#include <thrust/sort.h>

#define BLOCKSIZE 32

hpcparallel::smoothing::smoothing(int resolution, int binsize, int* bins, int filtersize) : resolution(resolution), binsize(binsize), bins(bins), filtersize(filtersize)
{
	window = new int[filtersize * filtersize];
	memset(window, 0, filtersize * filtersize * sizeof(int));
	filteredBins = new int[binsize];
	//memset(filteredBins, 0, binsize * sizeof(int));


}

__global__ void medianFilter3x3Kernel(const int* dev_bins, int* dev_filteredBins, int resolution, int binsize, int filtersize, int halfFS, int windowsize)
{
	int tx = blockDim.x * blockIdx.x + threadIdx.x;
	int ty = blockDim.y * blockIdx.y + threadIdx.y;
	int tlx = threadIdx.x;
	int tly = threadIdx.y;

	if (tx < resolution && ty < resolution)
	{
		//edge values for shared memory are values needed by the window but whose median is not calculated in this block.
		//shared array is 1 row/column bigger on every side for edge cases.
		__shared__ int sm_bins[BLOCKSIZE + 2][BLOCKSIZE + 2];

		//populate shared memory block values which are not edge values
		sm_bins[tly + 1][tlx + 1] = dev_bins[ty*resolution + tx];

		//check if index on the edge of the block
		bool tx_left_edge = (tlx == 0);
		bool ty_top_edge = (tly == 0);
		bool tx_right_edge = (tlx == BLOCKSIZE - 1);
		bool ty_bot_edge = (tly == BLOCKSIZE - 1);

		// -1 padding for values not in dev_bins
		if (tx_left_edge)
			sm_bins[tly + 1][tlx] = -1;
		else if (tx_right_edge)
			sm_bins[tly + 1][tlx + 2] = -1;
		if (ty_top_edge)
		{
			sm_bins[tly][tlx + 1] = -1;
			if (tx_left_edge)
				sm_bins[tly][tlx] = -1;
			else if (tx_right_edge)
				sm_bins[tly][tlx + 2] = -1;

		}
		else if (ty_bot_edge)
		{
			sm_bins[tly + 2][tlx + 1] = -1;
			if (tx_left_edge)
				sm_bins[tly + 2][tlx] = -1;
			else if (tx_right_edge)
				sm_bins[tly + 2][tlx + 2] = -1;
		}


		//check if shared memory edge is global edge and then don't include it
		tx_left_edge &= (tx > 0);
		tx_right_edge &= (tx < resolution - 1);
		ty_top_edge &= (ty > 0);
		ty_bot_edge &= (ty < resolution - 1);

		// pull edge values into shared memory using threads on edges of block
		if (tx_left_edge)
			sm_bins[tly + 1][tlx] = dev_bins[ty*resolution + tx - 1];
		else if (tx_right_edge)
			sm_bins[tly + 1][tlx + 2] = dev_bins[ty*resolution + tx + 1];
		if (ty_top_edge)
		{
			sm_bins[tly][tlx + 1] = dev_bins[(ty - 1)*resolution + tx];
			if (tx_left_edge)
				sm_bins[tly][tlx] = dev_bins[(ty - 1)*resolution + tx - 1];
			else if (tx_right_edge)
				sm_bins[tly][tlx + 2] = dev_bins[(ty - 1)*resolution + tx + 1];

		}
		else if (ty_bot_edge)
		{
			sm_bins[tly + 2][tlx + 1] = dev_bins[(ty + 1)*resolution + tx];
			if (tx_left_edge)
				sm_bins[tly + 2][tlx] = dev_bins[(ty + 1)*resolution + tx - 1];
			else if (tx_right_edge)
				sm_bins[tly + 2][tlx + 2] = dev_bins[(ty + 1)*resolution + tx + 1];
		}

		__syncthreads();

		int window[9] =
		{
			sm_bins[tly][tlx], sm_bins[tly][tlx + 1], sm_bins[tly][tlx + 2],
			sm_bins[tly + 1][tlx], sm_bins[tly + 1][tlx + 1], sm_bins[tly + 1][tlx + 2],
			sm_bins[tly + 2][tlx], sm_bins[tly + 2][tlx + 1], sm_bins[tly + 2][tlx + 2]
		};

	
		//thrust::sort(thrust::seq, window, window + 9);

		//bubble sort
		for (int i = 0; i < 9; ++i){
		
			for (int j = i + 1; j < 9; ++j){
				if (window[i] > window[j])
				{
					int temp = window[i];
					window[i] = window[j];
					window[j] = temp;
				}
			}
		}
		
		int edges = 0; // count number of elements equal to -1, values which are not in the global array
		for (int i = 0; i < 9; ++i)
		{
			if (window[i] == -1)
				edges++;
		}

		// the median is calculated only with values actually inside the array, values outside array are -1
		// for even number of values in window calculate average between the 2 middle values
		int median;
		int mi = (9 - edges);
		if (mi % 2 == 0)
		{
			//throwing away decimal part of average
			median = ((window[(mi / 2) - 1 + edges] + window[(mi / 2) + edges]) / 2);
		}
		else
		{
			median = window[(mi / 2) + edges];
		}
		
			
		dev_filteredBins[ty * resolution + tx] = median;
	}
}

template<int WINDOWSIZE, int FILTERSIZE, int EDGESIZE>
__global__ void medianFilterTemplateKernel(const int* dev_bins, int* dev_filteredBins, int resolution, int binsize)
{
	int tx = blockDim.x * blockIdx.x + threadIdx.x;
	int ty = blockDim.y * blockIdx.y + threadIdx.y;
	int tlx = threadIdx.x;
	int tly = threadIdx.y;

	// edge size is Filtersize/2 integer division, half the length of the window excluding the value we calculating in the middle
	
	if (tx < resolution && ty < resolution)
	{
		__shared__ int sm_bins[BLOCKSIZE + FILTERSIZE - 1][BLOCKSIZE + FILTERSIZE - 1];

		//populate shared memory
		//first populate values which fall inside the block (not edge values)
		sm_bins[tly + EDGESIZE][tlx + EDGESIZE] = dev_bins[ty*resolution + tx];

		//check if index on the edge of the block
		// in this case the edge of the block is the border of the block with thickness equal to edgesize
		bool tx_left_edge = (tlx < EDGESIZE);
		bool ty_top_edge = (tly < EDGESIZE);
		bool tx_right_edge = (tlx > BLOCKSIZE - EDGESIZE - 1);
		bool ty_bot_edge = (tly > BLOCKSIZE - EDGESIZE - 1);

		// -1 padding for edge values
		if (tx_left_edge)
			sm_bins[tly + EDGESIZE][tlx] = -1;
		else if (tx_right_edge)
			sm_bins[tly + EDGESIZE][tlx + (2 * EDGESIZE)] = -1;
		if (ty_top_edge)
		{
			sm_bins[tly][tlx + EDGESIZE] = -1;
			if (tx_left_edge)
				sm_bins[tly][tlx] = -1;
			else if (tx_right_edge)
				sm_bins[tly][tlx + (2 * EDGESIZE)] = -1;

		}
		else if (ty_bot_edge)
		{
			sm_bins[tly + (2 * EDGESIZE)][tlx + EDGESIZE] = -1;
			if (tx_left_edge)
				sm_bins[tly + (2 * EDGESIZE)][tlx] = -1;
			else if (tx_right_edge)
				sm_bins[tly + (2 * EDGESIZE)][tlx + (2 * EDGESIZE)] = -1;
		}

		//check if shared memory edge is global edge and then don't include it
		tx_left_edge &= (tx - EDGESIZE  > 0);
		tx_right_edge &= (tx + EDGESIZE < resolution);
		ty_top_edge &= (ty - EDGESIZE > 0);
		ty_bot_edge &= (ty + EDGESIZE < resolution);

		//populate edge values
		if (tx_left_edge)
			sm_bins[tly + EDGESIZE][tlx] = dev_bins[ty*resolution + tx - EDGESIZE];
		else if (tx_right_edge)
			sm_bins[tly + EDGESIZE][tlx + (2 * EDGESIZE)] = dev_bins[ty*resolution + tx + EDGESIZE];
		if (ty_top_edge)
		{
			sm_bins[tly][tlx + EDGESIZE] = dev_bins[(ty - EDGESIZE)*resolution + tx];
			if (tx_left_edge)
				sm_bins[tly][tlx] = dev_bins[(ty - EDGESIZE)*resolution + tx - EDGESIZE];
			else if (tx_right_edge)
				sm_bins[tly][tlx + (2 * EDGESIZE)] = dev_bins[(ty - EDGESIZE)*resolution + tx + EDGESIZE];

		}
		else if (ty_bot_edge)
		{
			sm_bins[tly + (2 * EDGESIZE)][tlx + EDGESIZE] = dev_bins[(ty + EDGESIZE)*resolution + tx];
			if (tx_left_edge)
				sm_bins[tly + (2 * EDGESIZE)][tlx] = dev_bins[(ty + EDGESIZE)*resolution + tx - EDGESIZE];
			else if (tx_right_edge)
				sm_bins[tly + (2 * EDGESIZE)][tlx + (2 * EDGESIZE)] = dev_bins[(ty + EDGESIZE)*resolution + tx + EDGESIZE];
		}

		__syncthreads();

		

		// create window for this thread index
		int window[WINDOWSIZE];

		int startx = tlx;
		int starty = tly;
		int endx = tlx + FILTERSIZE;
		int endy = tly + FILTERSIZE;

		int i = 0;
		for (int y = starty; y < endy; ++y)
		{
			for (int x = startx; x < endx; ++x)
			{
				window[i] = sm_bins[y][x];
				i++;
				
			}
		}

		/*if (tx == 0 && ty == 0)
		{
			printf("0x0y\n%d %d %d %d %d\n%d %d %d %d %d\n%d %d %d %d %d\n%d %d %d %d %d\n%d %d %d %d %d\n\n", window[0], window[1], window[2], window[3], window[4], window[5], window[6], window[7], window[8], window[9], window[10], window[11], window[12], window[13], window[14], window[15], window[16], window[17], window[18], window[19], window[20], window[21], window[22], window[23], window[24]);
		}
		if (tx == 31 && ty == 0)
		{
			printf("31x0y\n%d %d %d %d %d\n%d %d %d %d %d\n%d %d %d %d %d\n%d %d %d %d %d\n%d %d %d %d %d\n\n", window[0], window[1], window[2], window[3], window[4], window[5], window[6], window[7], window[8], window[9], window[10], window[11], window[12], window[13], window[14], window[15], window[16], window[17], window[18], window[19], window[20], window[21], window[22], window[23], window[24]);
		}
		if (tx == 0 && ty == 31)
		{
			printf("0x31y\n%d %d %d %d %d\n%d %d %d %d %d\n%d %d %d %d %d\n%d %d %d %d %d\n%d %d %d %d %d\n\n", window[0], window[1], window[2], window[3], window[4], window[5], window[6], window[7], window[8], window[9], window[10], window[11], window[12], window[13], window[14], window[15], window[16], window[17], window[18], window[19], window[20], window[21], window[22], window[23], window[24]);
		}
		if (tx == 31 && ty == 31)
		{
			printf("31x31y\n%d %d %d %d %d\n%d %d %d %d %d\n%d %d %d %d %d\n%d %d %d %d %d\n%d %d %d %d %d\n\n", window[0], window[1], window[2], window[3], window[4], window[5], window[6], window[7], window[8], window[9], window[10], window[11], window[12], window[13], window[14], window[15], window[16], window[17], window[18], window[19], window[20], window[21], window[22], window[23], window[24]);
		}*/

		//sort window
		thrust::sort(thrust::seq, window, window + WINDOWSIZE);

		int edges = 0; // count number of elements equal to -1, values which are not in the global array

		for (int i = 0; i < WINDOWSIZE; ++i)
		{
			if (window[i] == -1)
				edges++;
		}
		// the median is calculated only with values actually inside the array, values outside array are -1
		// for even number of values in window calculate average between the 2 middle values
		int median;
		int mi = (WINDOWSIZE - edges);

		if (mi % 2 == 0)
		{
			//throwing away decimal part of average
			median = ((window[(mi / 2) - 1 + edges] + window[(mi / 2) + edges]) / 2);
		}
		else
		{
			median = window[(mi / 2) + edges];
		}

		dev_filteredBins[ty * resolution + tx] = median;

	}
}


#define cudaSafe(statuscode, description) { gpuAssert(statuscode, description, __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, char* description, char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
	  std::cout <<"Cuda error: " << description << ", " << hipGetErrorString(code) << " " << file << " " << line << std::endl;;
	 // if (abort) exit(code);
   }
}

int* hpcparallel::smoothing::applyFilter()
{
	//struct hipFuncAttributes funcAttrib;
	//cudaSafe(hipFuncGetAttributes(&funcAttrib, reinterpret_cast<const void*>(medianFilterTemplateKernel<20>)), "cudafuncgetattributes");
	//printf("%s numRegs=%d\n", "medianFilterTemplateKernel", funcAttrib.numRegs);
	int* dev_bins = 0;
	int* dev_filteredBins = 0;
	cudaMedianFilter(dev_bins, dev_filteredBins);
	cudaSafe(hipDeviceReset(), "cuda device reset");
	
	return filteredBins;
}

void hpcparallel::smoothing::cudaMedianFilter(int* dev_bins, int* dev_filteredBins)
{

	cudaSafe(hipSetDevice(0), "set device");

	cudaSafe(hipMalloc((void**)&dev_bins, binsize * sizeof(int)), "cuda malloc dev_bins");
	cudaSafe(hipMalloc((void**)&dev_filteredBins, binsize * sizeof(int)), "cuda malloc dev_filteredBins");

	cudaSafe(hipMemcpy(dev_bins, bins, binsize * sizeof(int), hipMemcpyHostToDevice), "cuda memcpy htd dev_bins");
	cudaSafe(hipMemset(dev_filteredBins, 0, binsize * sizeof(int)), "cuda memset dev_filteredBins");
	

	dim3 numThreads(BLOCKSIZE, BLOCKSIZE);
	dim3 numBlocks;
	numBlocks.x = (int)ceil(resolution / (float)numThreads.x);
	numBlocks.y = (int)ceil(resolution / (float)numThreads.y);

	//std::cout << "numBlocks & numThreads: " << numBlocks.x << " " << numThreads.x << " " << numBlocks.x * numThreads.x << " " << resolution << std::endl;
	//std::cout << "numBlocks & numThreads: " << numBlocks.y << " " << numThreads.y << " " << numBlocks.y * numThreads.y << " " << resolution << std::endl;

	//medianFilterKernel << <numBlocks, numThreads >> >(dev_bins, dev_filteredBins, resolution, binsize, filtersize, int(filtersize / 2), filtersize*filtersize);
	
	switch (filtersize)
	{
		case 3:
			medianFilter3x3Kernel << <numBlocks, numThreads >> >(dev_bins, dev_filteredBins, resolution, binsize, 3, 1, 9);
			break;
		case 5:
			medianFilterTemplateKernel<25, 5, 2> << <numBlocks, numThreads >> >(dev_bins, dev_filteredBins, resolution, binsize);
			break;
		case 7:
			medianFilterTemplateKernel<49, 7, 3> << <numBlocks, numThreads >> >(dev_bins, dev_filteredBins, resolution, binsize);
			break;
		case 15:
			medianFilterTemplateKernel<225, 15, 7> << <numBlocks, numThreads >> >(dev_bins, dev_filteredBins, resolution, binsize);
			break;
		case 17:
			medianFilterTemplateKernel<289, 17, 8> << <numBlocks, numThreads >> >(dev_bins, dev_filteredBins, resolution, binsize);
			break;
		case 19:
			medianFilterTemplateKernel<361, 19, 9> << <numBlocks, numThreads >> >(dev_bins, dev_filteredBins, resolution, binsize);
			break;
		case 21:
			medianFilterTemplateKernel<441, 21, 10> << <numBlocks, numThreads >> >(dev_bins, dev_filteredBins, resolution, binsize);
			break;
	}
	

	cudaSafe(hipGetLastError(), "cuda launch");
	
	cudaSafe(hipDeviceSynchronize(), "cuda device synchronize");

	cudaSafe(hipMemcpy(filteredBins, dev_filteredBins, binsize * sizeof(int), hipMemcpyDeviceToHost), "cuda memcpy dth dev_filteredBins");
	
	cudaSafe(hipFree(dev_bins), "cuda free");
	cudaSafe(hipFree(dev_filteredBins), "cuda free");
}


hpcparallel::smoothing::~smoothing()
{
	delete window;
	delete filteredBins;
}
