#include "hip/hip_runtime.h"
#include "smoothing.h"
#include <iostream>
#include <algorithm>
#include <math.h> 
#include <cstring>
#include "hip/hip_runtime.h"
#include ""
#include <thrust/sort.h>

#define BLOCKSIZE 32

hpcparallel::smoothing::smoothing(int resolution, int binsize, int* bins, int filtersize) : resolution(resolution), binsize(binsize), bins(bins), filtersize(filtersize)
{
	window = new int[filtersize * filtersize];
	memset(window, 0, filtersize * filtersize * sizeof(int));
	filteredBins = new int[binsize];
	//memset(filteredBins, 0, binsize * sizeof(int));


}

__global__ void medianFilterKernel(const int* dev_bins, int* dev_filteredBins, int resolution, int binsize, int filtersize, int halfFS, int windowsize)
{
	int tx = blockDim.x * blockIdx.x + threadIdx.x;
	int ty = blockDim.y * blockIdx.y + threadIdx.y;

	int startx = tx - halfFS;
	int starty = ty - halfFS;
	int endx = startx + filtersize;
	int endy = starty + filtersize;

	if (tx < resolution && ty < resolution)
	{
		//__shared__ int* sbins = new int[(blockDim.y + halfFS)*(blockDim.x + halfFS)];

		int* window = new int[windowsize];

		int i = 0;
		for (int y = starty; y < endy; ++y)
		{
			for (int x = startx; x < endx; ++x)
			{
				if (y >= 0 && y < resolution && x >= 0 && x < resolution)
				{
					window[i] = dev_bins[y*resolution + x];
					++i;
				}
			}
		}

		thrust::sort(thrust::seq, window, window + i);

		int median;
		if (i % 2 == 0)
		{
			//throwing away decimal part of average
			median = (window[(i / 2) - 1] + window[(i / 2)]) / 2;
		}
		else
		{
			median = window[(i / 2)];
		}

		dev_filteredBins[ty * resolution + tx] = median;

	}
}

#define cudaSafe(statuscode, description) { gpuAssert(statuscode, description, __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, char* description, char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
	  std::cout <<"Cuda error: " << description << ", " << hipGetErrorString(code) << " " << file << " " << line << std::endl;;
	 // if (abort) exit(code);
   }
}

int* hpcparallel::smoothing::applyFilter()
{
	//for (int i = 0; i < binsize; ++i)
	//{
	//	//std::cout << i << std::endl;
	//	filteredBins[i] = median(i);
	//}
	int* dev_bins = 0;
	int* dev_filteredBins = 0;
	cudaMedianFilter(dev_bins, dev_filteredBins);
	cudaSafe(hipDeviceReset(), "cuda device reset");
	
	return filteredBins;
}

void hpcparallel::smoothing::cudaMedianFilter(int* dev_bins, int* dev_filteredBins)
{

	cudaSafe(hipSetDevice(0), "set device");

	cudaSafe(hipMalloc((void**)&dev_bins, binsize * sizeof(int)), "cuda malloc dev_bins");
	cudaSafe(hipMalloc((void**)&dev_filteredBins, binsize * sizeof(int)), "cuda malloc dev_filteredBins");

	cudaSafe(hipMemcpy(dev_bins, bins, binsize * sizeof(int), hipMemcpyHostToDevice), "cuda memcpy htd dev_bins");
	cudaSafe(hipMemset(dev_filteredBins, 0, binsize * sizeof(int)), "cuda memset dev_filteredBins");
	

	dim3 numThreads(BLOCKSIZE, BLOCKSIZE);
	dim3 numBlocks;
	numBlocks.x = (int)ceil(resolution / (float)numThreads.x);
	numBlocks.y = (int)ceil(resolution / (float)numThreads.y);

	//std::cout << "numBlocks & numThreads: " << numBlocks.x << " " << numThreads.x << " " << numBlocks.x * numThreads.x << " " << resolution << std::endl;
	//std::cout << "numBlocks & numThreads: " << numBlocks.y << " " << numThreads.y << " " << numBlocks.y * numThreads.y << " " << resolution << std::endl;

	medianFilterKernel << <numBlocks, numThreads >> >(dev_bins, dev_filteredBins, resolution, binsize, filtersize, int(filtersize / 2), filtersize*filtersize);

	cudaSafe(hipGetLastError(), "cuda launch");
	
	cudaSafe(hipDeviceSynchronize(), "cuda device synchronize");

	cudaSafe(hipMemcpy(filteredBins, dev_filteredBins, binsize * sizeof(int), hipMemcpyDeviceToHost), "cuda memcpy dth dev_filteredBins");
	
	cudaSafe(hipFree(dev_bins), "cuda free");
	cudaSafe(hipFree(dev_filteredBins), "cuda free");
}



int hpcparallel::smoothing::median(int currentBin)
{

	int binx = currentBin % resolution;
	int biny = currentBin / resolution;
	int halfFS = int(filtersize / 2);
	int startx = binx - halfFS;
	int starty = biny - halfFS;
	int endx = startx + filtersize;
	int endy = starty + filtersize;

	//std::cout << "binxy"<< binx << " " << biny << std::endl;

	//std::cout << startx << " " << endx << " " << starty << " " <<endy << std::endl;




	int i = 0;
	for (int y = starty; y < endy; ++y)
	{
		for (int x = startx; x < endx; ++x)
		{
			if (y >= 0 && y < resolution && x >= 0 && x < resolution)
			{
				window[i] = bins[y*resolution + x];
				++i;
			}
		}

	}


	std::sort(window, window + i);

	/*for (int j = 0; j < i; ++j)
	{
	if (j % filtersize == 0)
	{
	std::cout << std::endl;
	}
	std::cout << window[j] << " ";
	}*/


	int median;
	if (i % 2 == 0)
	{
		//throwing away decimal part of average
		median = (window[(i / 2) - 1] + window[(i / 2)]) / 2;
	}
	else
	{
		median = window[(i / 2)];
	}
	return median;
}



hpcparallel::smoothing::~smoothing()
{
	delete window;
	delete filteredBins;
}
