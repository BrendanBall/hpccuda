#include "hip/hip_runtime.h"
#include "smoothing.h"
#include <iostream>
#include <algorithm>
#include <math.h> 
#include <cstring>
#include "hip/hip_runtime.h"
#include "hip/device_functions.h"
#include ""
#include <thrust/sort.h>

#define BLOCKSIZE 32

hpcparallel::smoothing::smoothing(int resolution, int binsize, int* bins, int filtersize) : resolution(resolution), binsize(binsize), bins(bins), filtersize(filtersize)
{
	window = new int[filtersize * filtersize];
	memset(window, 0, filtersize * filtersize * sizeof(int));
	filteredBins = new int[binsize];
	//memset(filteredBins, 0, binsize * sizeof(int));


}

__global__ void medianFilter3x3Kernel(const int* dev_bins, int* dev_filteredBins, int resolution, int binsize, int filtersize, int halfFS, int windowsize)
{
	int tx = blockDim.x * blockIdx.x + threadIdx.x;
	int ty = blockDim.y * blockIdx.y + threadIdx.y;
	int tlx = threadIdx.x;
	int tly = threadIdx.y;

	if (tx < resolution && ty < resolution)
	{
		//edge values for shared memory are values needed by the window but whose median is not calculated in this block.
		//shared array is 1 row/column bigger on every side for edge cases.
		__shared__ int sm_bins[BLOCKSIZE + 2][BLOCKSIZE + 2];

		//populate shared memory block
		sm_bins[threadIdx.y + 1][threadIdx.x + 1] = dev_bins[ty*resolution + tx];

		//get edge values needed for sliding window
		bool tx_left_edge = (tlx == 0);
		bool ty_top_edge = (tly == 0);
		bool tx_right_edge = (tlx == BLOCKSIZE - 1);
		bool ty_bot_edge = (tly == BLOCKSIZE - 1);

		// -1 padding for values not in dev_bins
		if (tx_left_edge)
			sm_bins[tly + 1][tlx] = -1;
		else if (tx_right_edge)
			sm_bins[tly + 1][tlx + 2] = -1;
		if (ty_top_edge)
		{
			sm_bins[tly][tlx + 1] = -1;
			if (tx_left_edge)
				sm_bins[tly][tlx] = -1;
			else if (tx_right_edge)
				sm_bins[tly][tlx + 2] = -1;

		}
		else if (ty_bot_edge)
		{
			sm_bins[tly + 2][tlx + 1] = -1;
			if (tx_left_edge)
				sm_bins[tly + 2][tlx] = -1;
			else if (tx_right_edge)
				sm_bins[tly + 2][tlx + 2] = -1;
		}


		//check if shared memory edge is global edge and then don't include it
		tx_left_edge &= (tx > 0);
		tx_right_edge &= (tx < resolution - 1);
		ty_top_edge &= (ty > 0);
		ty_bot_edge &= (ty < resolution - 1);

		// pull edge values into shared memory
		if (tx_left_edge)
			sm_bins[tly + 1][tlx] = dev_bins[ty*resolution + tx - 1];
		else if (tx_right_edge)
			sm_bins[tly + 1][tlx + 2] = dev_bins[ty*resolution + tx + 1];
		if (ty_top_edge)
		{
			sm_bins[tly][tlx + 1] = dev_bins[(ty - 1)*resolution + tx];
			if (tx_left_edge)
				sm_bins[tly][tlx] = dev_bins[(ty - 1)*resolution + tx - 1];
			else if (tx_right_edge)
				sm_bins[tly][tlx + 2] = dev_bins[(ty - 1)*resolution + tx + 1];

		}
		else if (ty_bot_edge)
		{
			sm_bins[tly + 2][tlx + 1] = dev_bins[(ty + 1)*resolution + tx];
			if (tx_left_edge)
				sm_bins[tly + 2][tlx] = dev_bins[(ty + 1)*resolution + tx - 1];
			else if (tx_right_edge)
				sm_bins[tly + 2][tlx + 2] = dev_bins[(ty + 1)*resolution + tx + 1];
		}

		__syncthreads();

		int window[9] =
		{
			sm_bins[threadIdx.y][threadIdx.x], sm_bins[threadIdx.y][threadIdx.x + 1], sm_bins[threadIdx.y][threadIdx.x + 2],
			sm_bins[threadIdx.y + 1][threadIdx.x], sm_bins[threadIdx.y + 1][threadIdx.x + 1], sm_bins[threadIdx.y + 1][threadIdx.x + 2],
			sm_bins[threadIdx.y + 2][threadIdx.x], sm_bins[threadIdx.y + 2][threadIdx.x + 1], sm_bins[threadIdx.y + 2][threadIdx.x + 2]
		};

	
		//thrust::sort(thrust::seq, window, window + 9);

		//bubble sort
		for (int i = 0; i < 9; ++i){
		
			for (int j = i + 1; j < 9; ++j){
				if (window[i] > window[j])
				{
					int temp = window[i];
					window[i] = window[j];
					window[j] = temp;
				}
			}
		}
		
		int edges = 0; // count number of elements equal to -1 which are not in the global array
		for (int i = 0; i < 9; ++i)
		{
			if (window[i] == -1)
				edges++;
		}

		int median;
		int i = (9 - edges);
		if (i % 2 == 0)
		{
			//throwing away decimal part of average
			median = ((window[(i / 2) - 1 + edges] + window[(i / 2) + edges]) / 2);
		}
		else
		{
			median = window[(i / 2) + edges];
		}
		
			
		dev_filteredBins[ty * resolution + tx] = median;
	}
}

template<int WINDOWSIZE, int FILTERSIZE, int EDGESIZE>
__global__ void medianFilterTemplateKernel(const int* dev_bins, int* dev_filteredBins, int resolution, int binsize)
{
	int tx = blockDim.x * blockIdx.x + threadIdx.x;
	int ty = blockDim.y * blockIdx.y + threadIdx.y;

	// edge size is Filtersize/2 integer division, half the length of the window excluding the value we calculating
	int startx = tx - EDGESIZE;
	int starty = ty - EDGESIZE;
	int endx = startx + FILTERSIZE;
	int endy = starty + FILTERSIZE;


	if (tx < resolution && ty < resolution)
	{
		//__shared__ int* sbins = new int[(blockDim.y + halfFS)*(blockDim.x + halfFS)];

		int window[WINDOWSIZE];
		int edges = 0; // count number of elements equal to -1 which are not in the global array

		int i = 0;
		for (int y = starty; y < endy; ++y)
		{
			for (int x = startx; x < endx; ++x)
			{
				if (y >= 0 && y < resolution && x >= 0 && x < resolution)
				{
					window[i] = dev_bins[y*resolution + x];
					
				}
				else
				{
					window[i] = -1;
					edges++;
				}
				i++;
			}
		}

		thrust::sort(thrust::seq, window, window + WINDOWSIZE);
		int median;
		int mi = (WINDOWSIZE - edges);

		if (mi % 2 == 0)
		{
			//throwing away decimal part of average
			median = ((window[(mi / 2) - 1 + edges] + window[(mi / 2) + edges]) / 2);
		}
		else
		{
			median = window[(mi / 2) + edges];
		}

		dev_filteredBins[ty * resolution + tx] = median;

	}
}

//template __global__ void medianFilterTemplateKernel<4>(const int* dev_bins, int* dev_filteredBins, int resolution, int binsize, int filtersize, int halfFS, int windowsize);
__global__ void medianFilterKernel(const int* dev_bins, int* dev_filteredBins, int resolution, int binsize, int filtersize, int halfFS, int windowsize)
{
	int tx = blockDim.x * blockIdx.x + threadIdx.x;
	int ty = blockDim.y * blockIdx.y + threadIdx.y;

	int startx = tx - halfFS;
	int starty = ty - halfFS;
	int endx = startx + filtersize;
	int endy = starty + filtersize;


	if (tx < resolution && ty < resolution)
	{
		//__shared__ int* sbins = new int[(blockDim.y + halfFS)*(blockDim.x + halfFS)];

		int* window = new int[windowsize];

		int i = 0;
		for (int y = starty; y < endy; ++y)
		{
			for (int x = startx; x < endx; ++x)
			{
				if (y >= 0 && y < resolution && x >= 0 && x < resolution)
				{
					window[i] = dev_bins[y*resolution + x];
					++i;
				}
			}
		}

		thrust::sort(thrust::seq, window, window + i);

		int median;
		if (i % 2 == 0)
		{
			//throwing away decimal part of average
			median = (window[(i / 2) - 1] + window[(i / 2)]) / 2;
		}
		else
		{
			median = window[(i / 2)];
		}

		dev_filteredBins[ty * resolution + tx] = median;

	}
}

#define cudaSafe(statuscode, description) { gpuAssert(statuscode, description, __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, char* description, char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
	  std::cout <<"Cuda error: " << description << ", " << hipGetErrorString(code) << " " << file << " " << line << std::endl;;
	 // if (abort) exit(code);
   }
}

int* hpcparallel::smoothing::applyFilter()
{
	//struct hipFuncAttributes funcAttrib;
	//cudaSafe(hipFuncGetAttributes(&funcAttrib, reinterpret_cast<const void*>(medianFilterTemplateKernel<20>)), "cudafuncgetattributes");
	//printf("%s numRegs=%d\n", "medianFilterTemplateKernel", funcAttrib.numRegs);
	int* dev_bins = 0;
	int* dev_filteredBins = 0;
	cudaMedianFilter(dev_bins, dev_filteredBins);
	cudaSafe(hipDeviceReset(), "cuda device reset");
	
	return filteredBins;
}

void hpcparallel::smoothing::cudaMedianFilter(int* dev_bins, int* dev_filteredBins)
{

	cudaSafe(hipSetDevice(0), "set device");

	cudaSafe(hipMalloc((void**)&dev_bins, binsize * sizeof(int)), "cuda malloc dev_bins");
	cudaSafe(hipMalloc((void**)&dev_filteredBins, binsize * sizeof(int)), "cuda malloc dev_filteredBins");

	cudaSafe(hipMemcpy(dev_bins, bins, binsize * sizeof(int), hipMemcpyHostToDevice), "cuda memcpy htd dev_bins");
	cudaSafe(hipMemset(dev_filteredBins, 0, binsize * sizeof(int)), "cuda memset dev_filteredBins");
	

	dim3 numThreads(BLOCKSIZE, BLOCKSIZE);
	dim3 numBlocks;
	numBlocks.x = (int)ceil(resolution / (float)numThreads.x);
	numBlocks.y = (int)ceil(resolution / (float)numThreads.y);

	//std::cout << "numBlocks & numThreads: " << numBlocks.x << " " << numThreads.x << " " << numBlocks.x * numThreads.x << " " << resolution << std::endl;
	//std::cout << "numBlocks & numThreads: " << numBlocks.y << " " << numThreads.y << " " << numBlocks.y * numThreads.y << " " << resolution << std::endl;

	//medianFilterKernel << <numBlocks, numThreads >> >(dev_bins, dev_filteredBins, resolution, binsize, filtersize, int(filtersize / 2), filtersize*filtersize);
	
	switch (filtersize)
	{
		case 3:
			medianFilter3x3Kernel << <numBlocks, numThreads >> >(dev_bins, dev_filteredBins, resolution, binsize, 3, 1, 9);
			break;
		case 5:
			medianFilterTemplateKernel<25, 5, 2> << <numBlocks, numThreads >> >(dev_bins, dev_filteredBins, resolution, binsize);
			break;
		case 7:
			medianFilterTemplateKernel<49, 7, 3> << <numBlocks, numThreads >> >(dev_bins, dev_filteredBins, resolution, binsize);
			break;
		case 15:
			medianFilterTemplateKernel<225, 15, 7> << <numBlocks, numThreads >> >(dev_bins, dev_filteredBins, resolution, binsize);
			break;
		case 17:
			medianFilterTemplateKernel<289, 17, 8> << <numBlocks, numThreads >> >(dev_bins, dev_filteredBins, resolution, binsize);
			break;
		case 19:
			medianFilterTemplateKernel<361, 19, 9> << <numBlocks, numThreads >> >(dev_bins, dev_filteredBins, resolution, binsize);
			break;
		case 21:
			medianFilterTemplateKernel<441, 21, 10> << <numBlocks, numThreads >> >(dev_bins, dev_filteredBins, resolution, binsize);
			break;
	}
	

	cudaSafe(hipGetLastError(), "cuda launch");
	
	cudaSafe(hipDeviceSynchronize(), "cuda device synchronize");

	cudaSafe(hipMemcpy(filteredBins, dev_filteredBins, binsize * sizeof(int), hipMemcpyDeviceToHost), "cuda memcpy dth dev_filteredBins");
	
	cudaSafe(hipFree(dev_bins), "cuda free");
	cudaSafe(hipFree(dev_filteredBins), "cuda free");
}


hpcparallel::smoothing::~smoothing()
{
	delete window;
	delete filteredBins;
}
