#include "hip/hip_runtime.h"
#include "smoothing.h"
#include <iostream>
#include <algorithm>
#include <math.h> 
#include <cstring>
#include "hip/hip_runtime.h"
#include ""

#define BLOCKSIZE 32

hpcparallel::smoothing::smoothing(int resolution, int binsize, int* bins, int filtersize) : resolution(resolution), binsize(binsize), bins(bins), filtersize(filtersize)
{
	window = new int[filtersize * filtersize];
	memset(window, 0, filtersize * filtersize * sizeof(int));
	filteredBins = new int[binsize];
	//memset(filteredBins, 0, binsize * sizeof(int));


}

__global__ void medianFilterKernel(const int* dev_bins, int* dev_filteredBins, const int resolution, const int binsize, const int filtersize, const int windowsize)
{
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;

	if (x < resolution && y < resolution)
	{
		dev_filteredBins[y*resolution + x] = 2;
	}
}

int* hpcparallel::smoothing::applyFilter()
{
	//for (int i = 0; i < binsize; ++i)
	//{
	//	//std::cout << i << std::endl;
	//	filteredBins[i] = median(i);
	//}
	int* dev_bins = 0;
	int* dev_filteredBins = 0;
	cudaMedianFilter(dev_bins, dev_filteredBins);
	hipFree(dev_bins);
	hipFree(dev_filteredBins);
	hipError_t cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess)
	{
		std::cout << "hipDeviceReset failed" << std::endl;
	}
	return filteredBins;
}

hipError_t hpcparallel::smoothing::cudaMedianFilter(int* dev_bins, int* dev_filteredBins)
{
	

	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);

	if (cudaStatus != hipSuccess)
	{
		std::cout << "Cuda failed to set cuda device as 0" << std::endl;
		return cudaStatus;
	}

	cudaStatus = hipMalloc((void**)&dev_bins, binsize * sizeof(int));
	cudaStatus = hipMalloc((void**)&dev_filteredBins, binsize * sizeof(int));

	if (cudaStatus != hipSuccess)
	{
		std::cout << "hipMalloc failed" << std::endl;
		return cudaStatus;
	}

	cudaStatus = hipMemcpy(dev_bins, bins, binsize * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		std::cout << "hipMemcpy failed" << std::endl;
		return cudaStatus;
	}

	cudaStatus = hipMemset(dev_filteredBins, 0, binsize * sizeof(int));
	if (cudaStatus != hipSuccess)
	{
		std::cout << "hipMemset failed" << std::endl;
		return cudaStatus;
	}

	dim3 numThreads(BLOCKSIZE, BLOCKSIZE);
	dim3 numBlocks;
	numBlocks.x = (int)ceil(resolution / (float)numThreads.x);
	numBlocks.y = (int)ceil(resolution / (float)numThreads.y);

	//std::cout << "numBlocks & numThreads: " << numBlocks.x << " " << numThreads.x << " " << numBlocks.x * numThreads.x << " " << resolution << std::endl;
	//std::cout << "numBlocks & numThreads: " << numBlocks.y << " " << numThreads.y << " " << numBlocks.y * numThreads.y << " " << resolution << std::endl;

	medianFilterKernel<<<numBlocks, numThreads>>>(dev_bins, dev_filteredBins, resolution, binsize, filtersize, filtersize*filtersize);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		std::cout << "kernel launch failed: \n" << hipGetErrorString(cudaStatus) << std::endl;
		return cudaStatus;
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess)
	{
		std::cout << "hipDeviceSynchronize returned error: " << cudaStatus << std::endl;
		return cudaStatus;
	}

	cudaStatus = hipMemcpy(filteredBins, dev_filteredBins, binsize * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		std::cout << "hipMemcpy to host failed" << std::endl;
		return cudaStatus;
	}

	hipFree(dev_bins);
	hipFree(dev_filteredBins);
	return cudaStatus;
}



int hpcparallel::smoothing::median(int currentBin)
{

	int binx = currentBin % resolution;
	int biny = currentBin / resolution;
	int halfFS = int(filtersize / 2);
	int startx = binx - halfFS;
	int starty = biny - halfFS;
	int endx = startx + filtersize;
	int endy = starty + filtersize;

	//std::cout << "binxy"<< binx << " " << biny << std::endl;

	//std::cout << startx << " " << endx << " " << starty << " " <<endy << std::endl;




	int i = 0;
	for (int y = starty; y < endy; ++y)
	{
		for (int x = startx; x < endx; ++x)
		{
			if (y >= 0 && y < resolution && x >= 0 && x < resolution)
			{
				window[i] = bins[y*resolution + x];
				++i;
			}
		}

	}


	std::sort(window, window + i);

	/*for (int j = 0; j < i; ++j)
	{
	if (j % filtersize == 0)
	{
	std::cout << std::endl;
	}
	std::cout << window[j] << " ";
	}*/


	int median;
	if (i % 2 == 0)
	{
		//throwing away decimal part of average
		median = (window[(i / 2) - 1] + window[(i / 2)]) / 2;
	}
	else
	{
		median = window[(i / 2)];
	}
	return median;
}



hpcparallel::smoothing::~smoothing()
{
	delete window;
	delete filteredBins;
}
